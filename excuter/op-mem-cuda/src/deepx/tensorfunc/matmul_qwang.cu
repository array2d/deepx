#include "hip/hip_runtime.h"
#include "deepx/tensorfunc/cuda.hpp"

// #include <cuda_fp64.h>
// #include <cuda_fp32.h>
#include "deepx/tensor.hpp"

#include "deepx/tensorfunc/matmul_qwang.cuh"
#include "deepx/tensorfunc/authors.hpp"
#include "deepx/tensorfunc/cuda.hpp"

namespace deepx::tensorfunc
{

#define BLOCK_SIZE 32

    template <typename T>
    __global__ void matmul_kernel(T *C, const T *A, const T *B,
                                     int M, int N, int K)
    {
        // 定义共享内存块，用于缓存A和B的矩阵块
        __shared__ T tileA[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ T tileB[BLOCK_SIZE][BLOCK_SIZE];

        // 计算当前线程处理的全局矩阵位置
        int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
        int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

        T sum = 0.0;

        // 分块循环处理整个K维度
        for (int t = 0; t < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t)
        {
            // 计算当前块的起始位置
            int tiledK = t * BLOCK_SIZE;

            // 加载A的块到共享内存（行优先）
            int loadA_col = tiledK + threadIdx.x;
            if (row < M && loadA_col < K)
            {
                tileA[threadIdx.y][threadIdx.x] = A[row * K + loadA_col];
            }
            else
            {
                tileA[threadIdx.y][threadIdx.x] = 0.0; // 填充0处理边界
            }

            // 加载B的块到共享内存（列优先等效处理）
            int loadB_row = tiledK + threadIdx.y;
            if (col < N && loadB_row < K)
            {
                tileB[threadIdx.y][threadIdx.x] = B[loadB_row * N + col];
            }
            else
            {
                tileB[threadIdx.y][threadIdx.x] = 0.0; // 填充0处理边界
            }

            __syncthreads(); // 确保块加载完成

            // 计算当前块的矩阵乘法贡献
            for (int k = 0; k < BLOCK_SIZE; ++k)
            {
                sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
            }

            __syncthreads(); // 确保计算完成再加载下一块
        }

        // 只将有效范围内的结果写入全局内存
        if (row < M && col < N)
        {
            C[row * N + col] = sum;
        }
    }

    template __global__ void matmul_kernel<double>(double *C, const double *A, const double *B,
                                                     int M, int N, int K);
    template __global__ void matmul_kernel<float>(float *C, const float *A, const float *B,
                                                     int M, int N, int K);
    template __global__ void matmul_kernel<half>(half *C, const half *A, const half *B,
                                                     int M, int N, int K);
    template __global__ void matmul_kernel<hip_bfloat16>(hip_bfloat16 *C, const hip_bfloat16 *A, const hip_bfloat16 *B,
                                                     int M, int N, int K);
    template __global__ void matmul_kernel<int64_t>(int64_t *C, const int64_t *A, const int64_t *B,
                                                     int M, int N, int K);
    template __global__ void matmul_kernel<int32_t>(int32_t *C, const int32_t *A, const int32_t *B,
                                                     int M, int N, int K);
    template __global__ void matmul_kernel<int16_t>(int16_t *C, const int16_t *A, const int16_t *B,
                                                     int M, int N, int K);
    template __global__ void matmul_kernel<int8_t>(int8_t *C, const int8_t *A, const int8_t *B,
                                                     int M, int N, int K);
    // 主机函数调用内核
    template <typename T>
    void launch_matmul(T *d_C, const T *d_A, const T *d_B,
                    int M, int N, int K)
    {
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                     (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

        matmul_kernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, M, N, K);
    }
    template void launch_matmul<double>(double *d_C, const double *d_A, const double *d_B,
                                        int M, int N, int K);
    template void launch_matmul<float>(float *d_C, const float *d_A, const float *d_B,
                                        int M, int N, int K);
    template void launch_matmul<half>(half *d_C, const half *d_A, const half *d_B,
                                        int M, int N, int K);
    template void launch_matmul<hip_bfloat16>(hip_bfloat16 *d_C, const hip_bfloat16 *d_A,   const hip_bfloat16 *d_B,
                                        int M, int N, int K);
    template void launch_matmul<int64_t>(int64_t *d_C, const int64_t *d_A, const int64_t *d_B,
                                        int M, int N, int K);
    template void launch_matmul<int32_t>(int32_t *d_C, const int32_t *d_A, const int32_t *d_B,
                                        int M, int N, int K);
    template void launch_matmul<int16_t>(int16_t *d_C, const int16_t *d_A, const int16_t *d_B,
                                        int M, int N, int K);
    template void launch_matmul<int8_t>(int8_t *d_C, const int8_t *d_A, const int8_t *d_B,
                                        int M, int N, int K);
}
