#include "hip/hip_runtime.h"
#ifndef DEEPX_TENSORFUNC_ELEMENTWISE_MIAO_BYTE_SQRT_CU
#define DEEPX_TENSORFUNC_ELEMENTWISE_MIAO_BYTE_SQRT_CU

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include "deepx/tensorfunc/cuda.hpp"
#include "deepx/tensorfunc/authors.hpp"
#include <cuda/std/cmath>

namespace deepx::tensorfunc
{
    // sqrt
    template <typename T>
    __global__ void sqrt_kernel(const T *A, T *C, const int size);
    template <>
    __global__ void sqrt_kernel<double>(const double *A, double *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = sqrt(A[idx]);
        }
    }
    template <>
    __global__ void sqrt_kernel<float>(const float *A, float *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = sqrtf(A[idx]);
        }
    }

    template <>
    __global__ void sqrt_kernel<__half>(const __half *A, __half *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = hsqrt(A[idx]);
        }
    }
     template <>
    __global__ void sqrt_kernel<hip_bfloat16>(const hip_bfloat16 *A, hip_bfloat16 *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = hsqrt(A[idx]);
        }
    }
    template <typename T>
    void launch_sqrt(int numBlocks, int blockSize, const T *a, T *c, const int size)
    {
        sqrt_kernel<<<numBlocks, blockSize>>>(a, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("Failed to launch sqrt kernel: " +
                                     std::string(hipGetErrorString(err)));
        }
    }
    template void launch_sqrt<double>(int numBlocks, int blockSize, const double *a, double *c, const int size);
    template void launch_sqrt<float>(int numBlocks, int blockSize, const float *a, float *c, const int size);
    template void launch_sqrt<__half>(int numBlocks, int blockSize, const __half *a, __half *c, const int size);
    template void launch_sqrt<hip_bfloat16>(int numBlocks, int blockSize, const hip_bfloat16 *a, hip_bfloat16 *c, const int size);
    // pow
    template <typename T>
    __global__ void pow_kernel(const T *A, const T *B, T *C, const int size);
    template <>
    __global__ void pow_kernel<double>(const double *A, const double *B, double *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = pow(A[idx], B[idx]);
        }
    }
    template <>
    __global__ void pow_kernel<float>(const float *A, const float *B, float *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = powf(A[idx], B[idx]);
        }
    }

    template <typename T>
    void launch_pow(int numBlocks, int blockSize, const T *a, const T *b, T *c, const int size)
    {
        pow_kernel<<<numBlocks, blockSize>>>(a, b, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("Failed to launch pow kernel: " +
                                     std::string(hipGetErrorString(err)));
        }
    }
    template void launch_pow<double>(int numBlocks, int blockSize, const double *a, const double *b, double *c, const int size);
    template void launch_pow<float>(int numBlocks, int blockSize, const float *a, const float *b, float *c, const int size);

    // powscalar
    template <typename T>
    __global__ void powscalar_kernel(const T *A, const T scalar, T *C, const int size);
    template <>
    __global__ void powscalar_kernel<double>(const double *A, const double scalar, double *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = pow(A[idx], scalar);
        }
    }
    template <>
    __global__ void powscalar_kernel<float>(const float *A, const float scalar, float *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = powf(A[idx], scalar);
        }
    }
    template __global__ void powscalar_kernel<double>(const double *A, const double scalar, double *C, const int size);
    template __global__ void powscalar_kernel<float>(const float *A, const float scalar, float *C, const int size);

    template <typename T>
    void launch_powscalar(int numBlocks, int blockSize, const T *a, const T scalar, T *c, const int size)
    {
        powscalar_kernel<<<numBlocks, blockSize>>>(a, scalar, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("Failed to launch powscalar kernel: " +
                                     std::string(hipGetErrorString(err)));
        }
    }
    template void launch_powscalar<double>(int numBlocks, int blockSize, const double *a, const double scalar, double *c, const int size);
    template void launch_powscalar<float>(int numBlocks, int blockSize, const float *a, const float scalar, float *c, const int size);

    // log
    template <typename T>
    __global__ void log_kernel(const T *A, T *C, const int size);
    template <>
    __global__ void log_kernel<double>(const double *A, double *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = logf(A[idx]);
        }
    }
    template <>
    __global__ void log_kernel<float>(const float *A, float *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = logf(A[idx]);
        }
    }
    template <>
    __global__ void log_kernel<__half>(const __half *A, __half *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = hlog(A[idx]);
        }
    }
    template <>
    __global__ void log_kernel<hip_bfloat16>(const hip_bfloat16 *A, hip_bfloat16 *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {   
            C[idx] = hlog(A[idx]);
        }
    }
    
    template <typename T>
    void launch_log(int numBlocks, int blockSize, const T *a, T *c, const int size)
    {
        log_kernel<<<numBlocks, blockSize>>>(a, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("Failed to launch log kernel: " +
                                     std::string(hipGetErrorString(err)));
        }
    }
    template void launch_log<double>(int numBlocks, int blockSize, const double *a, double *c, const int size);
    template void launch_log<float>(int numBlocks, int blockSize, const float *a, float *c, const int size);
    template void launch_log<__half>(int numBlocks, int blockSize, const __half *a, __half *c, const int size);
    template void launch_log<hip_bfloat16>(int numBlocks, int blockSize, const hip_bfloat16 *a, hip_bfloat16 *c, const int size);
    // exp
    template <typename T>
    __global__ void exp_kernel(const T *A, T *C, const int size);
    template <>
    __global__ void exp_kernel<double>(const double *A, double *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = exp(A[idx]);
        }
    }
    template <>
    __global__ void exp_kernel<float>(const float *A, float *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = expf(A[idx]);
        }
    }

    template <>
    __global__ void exp_kernel<__half>(const __half *A, __half *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = hexp(A[idx]);
        }
    }
    template <>
    __global__ void exp_kernel<hip_bfloat16>(const hip_bfloat16 *A, hip_bfloat16 *C, const int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            C[idx] = hexp(A[idx]);
        }
    }

    template <typename T>
    void launch_exp(int numBlocks, int blockSize, const T *a, T *c, const int size)
    {
        exp_kernel<<<numBlocks, blockSize>>>(a, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("Failed to launch exp kernel: " +
                                     std::string(hipGetErrorString(err)));
        }
    }
    template void launch_exp<double>(int numBlocks, int blockSize, const double *a, double *c, const int size);
    template void launch_exp<float>(int numBlocks, int blockSize, const float *a, float *c, const int size);
    template void launch_exp<__half>(int numBlocks, int blockSize, const __half *a, __half *c, const int size);
    template void launch_exp<hip_bfloat16>(int numBlocks, int blockSize, const hip_bfloat16 *a, hip_bfloat16 *c, const int size);
}
#endif // DEEPX_TENSORFUNC_ELEMENTWISE_MIAO_BYTE_SQRT_CU
