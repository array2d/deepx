#include "hip/hip_runtime.h"
#ifndef DEEPX_TENSORFUNC_ELEMENTWISE_MIAOBYTE_BASIC_CU
#define DEEPX_TENSORFUNC_ELEMENTWISE_MIAOBYTE_BASIC_CU

#include "deepx/tensorfunc/cuda.hpp"
#include "deepx/tensorfunc/authors.hpp"

namespace deepx::tensorfunc
{
     template <typename T>
    __global__ void add_kernel(const T* A, const T* B, T* C,const int size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] + B[idx];
        }
    }
    template __global__ void add_kernel<double>(const double* A, const double* B, double* C,const int size);
    template __global__ void add_kernel<float>(const float* A, const float* B, float* C,const int size);
    template __global__ void add_kernel<half>(const half* A, const half* B, half* C,const int size);
    template __global__ void add_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16* B, hip_bfloat16* C,const int size);
    template __global__ void add_kernel<int64_t>(const int64_t* A, const int64_t* B, int64_t* C,const int size);
    template __global__ void add_kernel<int32_t>(const int32_t* A, const int32_t* B, int32_t* C,const int size);
    template __global__ void add_kernel<int16_t>(const int16_t* A, const int16_t* B, int16_t* C,const int size);
    template __global__ void add_kernel<int8_t>(const int8_t* A, const int8_t* B, int8_t* C,const int size);
    
    template <typename T>
    void launch_add(int numBlocks, int blockSize,const T*  a, const  T* b,  T* c,const int size)
    {
         // 启动kernel
            add_kernel<<<numBlocks, blockSize>>>(a, b, c, size);
            // 检查kernel执行是否成功
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                throw std::runtime_error("Failed to launch add kernel: " + 
                                       std::string(hipGetErrorString(err)));
            }
    }

    template void launch_add<double>(int numBlocks, int blockSize,const double*  a, const  double* b,  double* c,const int size);
    template void launch_add<float>(int numBlocks, int blockSize,const float*  a, const  float* b,  float* c,const int size);
    template void launch_add<half>(int numBlocks, int blockSize,const half*  a, const  half* b,  half* c,const int size);
    template void launch_add<hip_bfloat16>(int numBlocks, int blockSize,const hip_bfloat16*  a, const  hip_bfloat16* b,  hip_bfloat16* c,const int size);
    template void launch_add<int64_t>(int numBlocks, int blockSize,const int64_t*  a, const  int64_t* b,  int64_t* c,const int size);
    template void launch_add<int32_t>(int numBlocks, int blockSize, const int32_t*  a, const  int32_t* b,  int32_t* c,const int size);
    template void launch_add<int16_t>(int numBlocks, int blockSize, const int16_t*  a, const  int16_t* b,  int16_t* c,const int size);
    template void launch_add<int8_t>(int numBlocks, int blockSize, const int8_t*  a, const  int8_t* b,  int8_t* c,const int size);


    template <typename T>
    __global__ void addscalar_kernel(const T* A, const T scalar, T* C,const int size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] + scalar;
        }
    }   
    template __global__ void addscalar_kernel<double>(const double* A, const double scalar, double* C,const int size);   
    template __global__ void addscalar_kernel<float>(const float* A, const float scalar, float* C,const int size);
    template __global__ void addscalar_kernel<half>(const half* A, const half scalar, half* C,const int size);
    template __global__ void addscalar_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16 scalar, hip_bfloat16* C,const int size);
    template __global__ void addscalar_kernel<int64_t>(const int64_t* A, const int64_t scalar, int64_t* C,const int size);
    template __global__ void addscalar_kernel<int32_t>(const int32_t* A, const int32_t scalar, int32_t* C,const int size);
    template __global__ void addscalar_kernel<int16_t>(const int16_t* A, const int16_t scalar, int16_t* C,const int size);
    template __global__ void addscalar_kernel<int8_t>(const int8_t* A, const int8_t scalar, int8_t* C,const int size);
    
    template <typename T>
    void launch_addscalar(const int numBlocks, const int blockSize, const T* a, const T scalar, T* c, const int size) {
        addscalar_kernel<<<numBlocks, blockSize>>>(a, scalar, c, size);
    }   
    template void launch_addscalar<double>(const int numBlocks, const int blockSize, const double* a, const double scalar, double* c, const int size);
    template void launch_addscalar<float>(const int numBlocks, const int blockSize, const float* a, const float scalar, float* c, const int size);
    template void launch_addscalar<half>(const int numBlocks, const int blockSize, const half* a, const half scalar, half* c, const int size);
    template void launch_addscalar<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16* a, const hip_bfloat16 scalar, hip_bfloat16* c, const int size);
    template void launch_addscalar<int64_t>(const int numBlocks, const int blockSize, const int64_t* a, const int64_t scalar, int64_t* c, const int size);  
    template void launch_addscalar<int32_t>(const int numBlocks, const int blockSize, const int32_t* a, const int32_t scalar, int32_t* c, const int size);
    template void launch_addscalar<int16_t>(const int numBlocks, const int blockSize, const int16_t* a, const int16_t scalar, int16_t* c, const int size);
    template void launch_addscalar<int8_t>(const int numBlocks, const int blockSize, const int8_t* a, const int8_t scalar, int8_t* c, const int size);


    template <typename T>
    __global__ void sub_kernel(const T* A, const T* B, T* C,const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] - B[idx];
        }   
    }
    template __global__ void sub_kernel<double>(const double* A, const double* B, double* C, const int size);   
    template __global__ void sub_kernel<float>(const float* A, const float* B, float* C, const int size);
    template __global__ void sub_kernel<half>(const half* A, const half* B, half* C, const int size);
    template __global__ void sub_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16* B, hip_bfloat16* C, const int size);
    template __global__ void sub_kernel<int64_t>(const int64_t* A, const int64_t* B, int64_t* C, const int size);
    template __global__ void sub_kernel<int32_t>(const int32_t* A, const int32_t* B, int32_t* C, const int size);
    template __global__ void sub_kernel<int16_t>(const int16_t* A, const int16_t* B, int16_t* C, const int size);
    template __global__ void sub_kernel<int8_t>(const int8_t* A, const int8_t* B, int8_t* C, const int size);

    template <typename T>
    void launch_sub(const int numBlocks, const int blockSize, const T* a, const T* b, T* c, const int size) {
        sub_kernel<<<numBlocks, blockSize>>>(a, b, c, size);
    }
    template void launch_sub<double>(const int numBlocks, const int blockSize, const double* a, const double* b, double* c, const int size);
    template void launch_sub<float>(const int numBlocks, const int blockSize, const float* a, const float* b, float* c, const int size);
    template void launch_sub<half>(const int numBlocks, const int blockSize, const half* a, const half* b, half* c, const int size);
    template void launch_sub<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16* a, const hip_bfloat16* b, hip_bfloat16* c, const int size);
    template void launch_sub<int64_t>(const int numBlocks, const int blockSize, const int64_t* a, const int64_t* b, int64_t* c, const int size);
    template void launch_sub<int32_t>(const int numBlocks, const int blockSize, const int32_t* a, const int32_t* b, int32_t* c, const int size);
    template void launch_sub<int16_t>(const int numBlocks, const int blockSize, const int16_t* a, const int16_t* b, int16_t* c, const int size);
    template void launch_sub<int8_t>(const int numBlocks, const int blockSize, const int8_t* a, const int8_t* b, int8_t* c, const int size);    
    
    template <typename T>
    __global__ void subscalar_kernel(const T* A, const T scalar, T* C,const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] - scalar;
        }
    }   
    template __global__ void subscalar_kernel<double>(const double* A, const double scalar, double* C,const int size);
    template __global__ void subscalar_kernel<float>(const float* A, const float scalar, float* C,const int size);
    template __global__ void subscalar_kernel<half>(const half* A, const half scalar, half* C,const int size);
    template __global__ void subscalar_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16 scalar, hip_bfloat16* C,const int size);
    template __global__ void subscalar_kernel<int64_t>(const int64_t* A, const int64_t scalar, int64_t* C,const int size);  
    template __global__ void subscalar_kernel<int32_t>(const int32_t* A, const int32_t scalar, int32_t* C,const int size);  
    template __global__ void subscalar_kernel<int16_t>(const int16_t* A, const int16_t scalar, int16_t* C,const int size);  
    template __global__ void subscalar_kernel<int8_t>(const int8_t* A, const int8_t scalar, int8_t* C,const int size);  

    template <typename T>
    void launch_subscalar(const int numBlocks, const int blockSize, const T* a, const T scalar, T* c, const int size) { 
        subscalar_kernel<<<numBlocks, blockSize>>>(a, scalar, c, size);
    }
    template void launch_subscalar<double>(const int numBlocks, const int blockSize, const double* a, const double scalar, double* c, const int size);
    template void launch_subscalar<float>(const int numBlocks, const int blockSize, const float* a, const float scalar, float* c, const int size);
    template void launch_subscalar<half>(const int numBlocks, const int blockSize, const half* a, const half scalar, half* c, const int size);
    template void launch_subscalar<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16* a, const hip_bfloat16 scalar, hip_bfloat16* c, const int size);  
    template void launch_subscalar<int64_t>(const int numBlocks, const int blockSize, const int64_t* a, const int64_t scalar, int64_t* c, const int size);  
    template void launch_subscalar<int32_t>(const int numBlocks, const int blockSize, const int32_t* a, const int32_t scalar, int32_t* c, const int size);  
    template void launch_subscalar<int16_t>(const int numBlocks, const int blockSize, const int16_t* a, const int16_t scalar, int16_t* c, const int size);  
    template void launch_subscalar<int8_t>(const int numBlocks, const int blockSize, const int8_t* a, const int8_t scalar, int8_t* c, const int size);    
    
     template <typename T>
    __global__ void mul_kernel(const T* A, const T* B, T* C,const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] * B[idx];
        }
    }  
    template __global__ void mul_kernel<double>(const double* A, const double* B, double* C,const int size);
    template __global__ void mul_kernel<float>(const float* A, const float* B, float* C,const int size);
    template __global__ void mul_kernel<half>(const half* A, const half* B, half* C,const int size);
    template __global__ void mul_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16* B, hip_bfloat16* C,const int size);
    template __global__ void mul_kernel<int64_t>(const int64_t* A, const int64_t* B, int64_t* C,const int size);
    template __global__ void mul_kernel<int32_t>(const int32_t* A, const int32_t* B, int32_t* C,const int size);    
    template __global__ void mul_kernel<int16_t>(const int16_t* A, const int16_t* B, int16_t* C,const int size);
    template __global__ void mul_kernel<int8_t>(const int8_t* A, const int8_t* B, int8_t* C,const int size);

    template <typename T>
    void launch_mul(const int numBlocks, const int blockSize, const T* a, const T* b, T* c, const int size) {
        mul_kernel<<<numBlocks, blockSize>>>(a, b, c, size);
    }   
    template void launch_mul<double>(const int numBlocks, const int blockSize, const double* a, const double* b, double* c, const int size);    
    template void launch_mul<float>(const int numBlocks, const int blockSize, const float* a, const float* b, float* c, const int size);
    template void launch_mul<half>(const int numBlocks, const int blockSize, const half* a, const half* b, half* c, const int size);
    template void launch_mul<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16* a, const hip_bfloat16* b, hip_bfloat16* c, const int size);
    template void launch_mul<int64_t>(const int numBlocks, const int blockSize, const int64_t* a, const int64_t* b, int64_t* c, const int size);    
    template void launch_mul<int32_t>(const int numBlocks, const int blockSize, const int32_t* a, const int32_t* b, int32_t* c, const int size);    
    template void launch_mul<int16_t>(const int numBlocks, const int blockSize, const int16_t* a, const int16_t* b, int16_t* c, const int size);    
    template void launch_mul<int8_t>(const int numBlocks, const int blockSize, const int8_t* a, const int8_t* b, int8_t* c, const int size);    
    
    template <typename T>
    __global__ void mulscalar_kernel(const T* A, const T scalar, T* C,const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] * scalar;
        }
    }   
    template __global__ void mulscalar_kernel<double>(const double* A, const double scalar, double* C,const int size);
    template __global__ void mulscalar_kernel<float>(const float* A, const float scalar, float* C,const int size);
    template __global__ void mulscalar_kernel<half>(const half* A, const half scalar, half* C,const int size);
    template __global__ void mulscalar_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16 scalar, hip_bfloat16* C,const int size);  
    template __global__ void mulscalar_kernel<int64_t>(const int64_t* A, const int64_t scalar, int64_t* C,const int size);  
    template __global__ void mulscalar_kernel<int32_t>(const int32_t* A, const int32_t scalar, int32_t* C,const int size);  
    template __global__ void mulscalar_kernel<int16_t>(const int16_t* A, const int16_t scalar, int16_t* C,const int size);  
    template __global__ void mulscalar_kernel<int8_t>(const int8_t* A, const int8_t scalar, int8_t* C,const int size);  
    
    template <typename T>
    void launch_mulscalar(const int numBlocks, const int blockSize, const T* a, const T scalar, T* c, const int size) {
        mulscalar_kernel<<<numBlocks, blockSize>>>(a, scalar, c, size);
    }
    template void launch_mulscalar<double>(const int numBlocks, const int blockSize, const double* a, const double scalar, double* c, const int size);
    template void launch_mulscalar<float>(const int numBlocks, const int blockSize, const float* a, const float scalar, float* c, const int size);
    template void launch_mulscalar<half>(const int numBlocks, const int blockSize, const half* a, const half scalar, half* c, const int size);
    template void launch_mulscalar<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16* a, const hip_bfloat16 scalar, hip_bfloat16* c, const int size);
    template void launch_mulscalar<int64_t>(const int numBlocks, const int blockSize, const int64_t* a, const int64_t scalar, int64_t* c, const int size);
    template void launch_mulscalar<int32_t>(const int numBlocks, const int blockSize, const int32_t* a, const int32_t scalar, int32_t* c, const int size);  
    template void launch_mulscalar<int16_t>(const int numBlocks, const int blockSize, const int16_t* a, const int16_t scalar, int16_t* c, const int size);  
    template void launch_mulscalar<int8_t>(const int numBlocks, const int blockSize, const int8_t* a, const int8_t scalar, int8_t* c, const int size);  
    
    template <typename T>
    __global__ void div_kernel(const T* A, const T* B, T* C,const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] / B[idx];
        }
    }   
    template __global__ void div_kernel<double>(const double* A, const double* B, double* C,const int size);
    template __global__ void div_kernel<float>(const float* A, const float* B, float* C,const int size);
    template __global__ void div_kernel<half>(const half* A, const half* B, half* C,const int size);
    template __global__ void div_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16* B, hip_bfloat16* C,const int size);    
    template __global__ void div_kernel<int64_t>(const int64_t* A, const int64_t* B, int64_t* C,const int size);  
    template __global__ void div_kernel<int32_t>(const int32_t* A, const int32_t* B, int32_t* C,const int size);  
    template __global__ void div_kernel<int16_t>(const int16_t* A, const int16_t* B, int16_t* C,const int size);  
    template __global__ void div_kernel<int8_t>(const int8_t* A, const int8_t* B, int8_t* C,const int size);  
    
    template <typename T>
    void launch_div(const int numBlocks, const int blockSize, const T* a, const T* b, T* c, const int size) {
        div_kernel<<<numBlocks, blockSize>>>(a, b, c, size);
    }
    template void launch_div<double>(const int numBlocks, const int blockSize, const double* a, const double* b, double* c, const int size);
    template void launch_div<float>(const int numBlocks, const int blockSize, const float* a, const float* b, float* c, const int size);
    template void launch_div<half>(const int numBlocks, const int blockSize, const half* a, const half* b, half* c, const int size);
    template void launch_div<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16* a, const hip_bfloat16* b, hip_bfloat16* c, const int size);
    template void launch_div<int64_t>(const int numBlocks, const int blockSize, const int64_t* a, const int64_t* b, int64_t* c, const int size);
    template void launch_div<int32_t>(const int numBlocks, const int blockSize, const int32_t* a, const int32_t* b, int32_t* c, const int size);    
    template void launch_div<int16_t>(const int numBlocks, const int blockSize, const int16_t* a, const int16_t* b, int16_t* c, const int size);  
    template void launch_div<int8_t>(const int numBlocks, const int blockSize, const int8_t* a, const int8_t* b, int8_t* c, const int size);  
    
    template <typename T>
    __global__ void divscalar_kernel(const T* A, const T scalar, T* C,const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;    
        if (idx < size) {
            C[idx] = A[idx] / scalar;
        }
    }   
    template __global__ void divscalar_kernel<double>(const double* A, const double scalar, double* C,const int size);
    template __global__ void divscalar_kernel<float>(const float* A, const float scalar, float* C,const int size);
    template __global__ void divscalar_kernel<half>(const half* A, const half scalar, half* C,const int size);
    template __global__ void divscalar_kernel<hip_bfloat16>(const hip_bfloat16* A, const hip_bfloat16 scalar, hip_bfloat16* C,const int size);
    template __global__ void divscalar_kernel<int64_t>(const int64_t* A, const int64_t scalar, int64_t* C,const int size);
    template __global__ void divscalar_kernel<int32_t>(const int32_t* A, const int32_t scalar, int32_t* C,const int size);
    template __global__ void divscalar_kernel<int16_t>(const int16_t* A, const int16_t scalar, int16_t* C,const int size);
    template __global__ void divscalar_kernel<int8_t>(const int8_t* A, const int8_t scalar, int8_t* C,const int size);
    
    template <typename T>
    void launch_divscalar(const int numBlocks, const int blockSize, const T* a, const T scalar, T* c, const int size) {
        divscalar_kernel<<<numBlocks, blockSize>>>(a, scalar, c, size);
    }   
    template void launch_divscalar<double>(const int numBlocks, const int blockSize, const double* a, const double scalar, double* c, const int size);
    template void launch_divscalar<float>(const int numBlocks, const int blockSize, const float* a, const float scalar, float* c, const int size);
    template void launch_divscalar<half>(const int numBlocks, const int blockSize, const half* a, const half scalar, half* c, const int size);
    template void launch_divscalar<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16* a, const hip_bfloat16 scalar, hip_bfloat16* c, const int size);
    template void launch_divscalar<int64_t>(const int numBlocks, const int blockSize, const int64_t* a, const int64_t scalar, int64_t* c, const int size);  
    template void launch_divscalar<int32_t>(const int numBlocks, const int blockSize, const int32_t* a, const int32_t scalar, int32_t* c, const int size);  
    template void launch_divscalar<int16_t>(const int numBlocks, const int blockSize, const int16_t* a, const int16_t scalar, int16_t* c, const int size);  
    template void launch_divscalar<int8_t>(const int numBlocks, const int blockSize, const int8_t* a, const int8_t scalar, int8_t* c, const int size);    
    
    template <typename T>
    __global__ void rdivscalar_kernel(const T scalar, const T* A, T* C,const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = scalar / A[idx];
        }
    }      
    template __global__ void rdivscalar_kernel<double>(const double scalar, const double* A, double* C,const int size); 
    template __global__ void rdivscalar_kernel<float>(const float scalar, const float* A, float* C,const int size);
    template __global__ void rdivscalar_kernel<half>(const half scalar, const half* A, half* C,const int size);
    template __global__ void rdivscalar_kernel<hip_bfloat16>(const hip_bfloat16 scalar, const hip_bfloat16* A, hip_bfloat16* C,const int size);
    template __global__ void rdivscalar_kernel<int64_t>(const int64_t scalar, const int64_t* A, int64_t* C,const int size);
    template __global__ void rdivscalar_kernel<int32_t>(const int32_t scalar, const int32_t* A, int32_t* C,const int size); 
    template __global__ void rdivscalar_kernel<int16_t>(const int16_t scalar, const int16_t* A, int16_t* C,const int size);
    template __global__ void rdivscalar_kernel<int8_t>(const int8_t scalar, const int8_t* A, int8_t* C,const int size);
    
    template <typename T>
    void launch_rdivscalar(const int numBlocks, const int blockSize, const T scalar, const T* a, T* c, const int size) {
        rdivscalar_kernel<<<numBlocks, blockSize>>>(scalar, a, c, size);
    }   
    template void launch_rdivscalar<double>(const int numBlocks, const int blockSize, const double scalar, const double* a, double* c, const int size); 
    template void launch_rdivscalar<float>(const int numBlocks, const int blockSize, const float scalar, const float* a, float* c, const int size);
    template void launch_rdivscalar<half>(const int numBlocks, const int blockSize, const half scalar, const half* a, half* c, const int size);
    template void launch_rdivscalar<hip_bfloat16>(const int numBlocks, const int blockSize, const hip_bfloat16 scalar, const hip_bfloat16* a, hip_bfloat16* c, const int size);
    template void launch_rdivscalar<int64_t>(const int numBlocks, const int blockSize, const int64_t scalar, const int64_t* a, int64_t* c, const int size);
    template void launch_rdivscalar<int32_t>(const int numBlocks, const int blockSize, const int32_t scalar, const int32_t* a, int32_t* c, const int size);
    template void launch_rdivscalar<int16_t>(const int numBlocks, const int blockSize, const int16_t scalar, const int16_t* a, int16_t* c, const int size);
    template void launch_rdivscalar<int8_t>(const int numBlocks, const int blockSize, const int8_t scalar, const int8_t* a, int8_t* c, const int size);
    
 
}

#endif // DEEPX_TENSORFUNC_ELEMENTWISE_MIAOBYTE_BASIC_CU
