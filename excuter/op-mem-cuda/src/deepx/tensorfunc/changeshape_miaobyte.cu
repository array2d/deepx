#include "hip/hip_runtime.h"
#ifndef DEEPX_TENSORFUNC_CHANGESHAPE_MIAOBYTE_CU
#define DEEPX_TENSORFUNC_CHANGESHAPE_MIAOBYTE_CU

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include "deepx/tensorfunc/cuda.hpp"
#include "deepx/tensorfunc/authors.hpp"
#include "deepx/tensorfunc/tensor_cuda.cuh"
#include "deepx/tensorfunc/vector_cuda.cuh"
#include "deepx/shape_changeshape.hpp"

namespace deepx::tensorfunc
{
    // transpose
    template <int DIM, typename T>
    __global__ void transpose_kernel(const T *inputData,
                                     const int *inputStrides,
                                     T *outputData,
                                     const int *outputStrides,
                                     const int dim,
                                     const int len,
                                     const int *dimOrder)
    {
        const int grid_stride = gridDim.x * blockDim.x;
        int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
        for (; thread_id < len; thread_id += grid_stride)
        {
            int input_indices[DIM];

            // 计算当前线程需要处理的索引
            linearTo(inputStrides, dim, input_indices, thread_id);

            int output_indices[DIM];

            // 根据 dim_order 和输入输出的形状计算新索引
            reorder(input_indices, dimOrder, dim, output_indices);
            int inputIdx = linearAt(inputStrides, dim, input_indices);
            int outputIdx = linearAt(outputStrides, dim, output_indices);
            outputData[outputIdx] = inputData[inputIdx];
        }
    }

    template <typename T>
    void launch_transpose(const T *input,
                          const int *inputStrides,
                          T *output,
                          const int *outputStrides,
                          const int dim,
                          const int len,
                          const int *dimOrder)
    {
        cudaVector<int> strides_d(inputStrides, dim);
        cudaVector<int> newStrides_d(outputStrides, dim);
        cudaVector<int> dimOrder_d(dimOrder, dim);

        auto [numBlocks, blockSize] = BestDims(len);
        switch (dim)
        {
        case 1:
            transpose_kernel<1, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 2:
            transpose_kernel<2, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 3:
            transpose_kernel<3, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 4:
            transpose_kernel<4, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 5:
            transpose_kernel<5, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 6:
            transpose_kernel<6, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 7:
            transpose_kernel<7, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 8:
            transpose_kernel<8, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 9:
            transpose_kernel<9, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 10:
            transpose_kernel<10, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 11:
            transpose_kernel<11, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;
        case 12:
            transpose_kernel<12, T><<<numBlocks, blockSize>>>(input, strides_d.data, output, newStrides_d.data, dim, len, dimOrder_d.data);
            break;

        default:
            throw std::runtime_error("dimension large than " + std::to_string(MAX_DIM));
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("cuda error");
        }
    }

    template void launch_transpose<double>(const double *input, const int *inputStrides, double *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);
    template void launch_transpose<float>(const float *input, const int *inputStrides, float *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);
    template void launch_transpose<hip_bfloat16>(const hip_bfloat16 *input, const int *inputStrides, hip_bfloat16 *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);
    template void launch_transpose<__half>(const __half *input, const int *inputStrides, __half *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);
    template void launch_transpose<int64_t>(const int64_t *input, const int *inputStrides, int64_t *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);
    template void launch_transpose<int32_t>(const int32_t *input, const int *inputStrides, int32_t *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);
    template void launch_transpose<int16_t>(const int16_t *input, const int *inputStrides, int16_t *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);
    template void launch_transpose<int8_t>(const int8_t *input, const int *inputStrides, int8_t *output, const int *outputStrides, const int dim, const int len, const int *dimOrder);

    // concat
    template <int DIM, typename T>
    __global__ void concat_kernel(const T **tensorsData,
                                  const int *inputStrides,
                                  T *outputData,
                                  const int *outputStrides,
                                  const int dim,
                                  const int outputLen,
                                  const int axis,
                                  const int numTensors,
                                  const int *shapeAtAxis)
    {
        const int grid_stride = gridDim.x * blockDim.x;
        int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

        cudaVector<int> outputIndices(DIM);
        cudaVector<int> currentTensorIndices(DIM);
        for (; thread_id < outputLen; thread_id += grid_stride)
        {
            linearTo(outputStrides, dim, outputIndices.data, thread_id);
            int concatIdxResult = outputIndices[axis];
            int concatIdxCurrentTensor = concatIdxResult;
            int tensorIdx = 0;
            while (tensorIdx < numTensors)
            {
                if (concatIdxCurrentTensor < shapeAtAxis[tensorIdx])
                {
                    break;
                }
                else
                {
                    concatIdxCurrentTensor -= shapeAtAxis[tensorIdx];
                    tensorIdx++;
                }
            }
            currentTensorIndices.copyFromDevice(outputIndices.data, dim);
            currentTensorIndices[axis] = concatIdxCurrentTensor;

            int idxCurrentTensor = linearAt(inputStrides + tensorIdx * dim, dim, currentTensorIndices.data);

            int idx = linearAt(outputStrides, dim, outputIndices.data);
            outputData[idx] = tensorsData[tensorIdx][idxCurrentTensor];
        }
    }

    template <typename T>
    void launch_concat(
        const T **tensorsData,
        const int *inputStrides,
        T *outputData,
        const int *outputStrides,
        const int dim,
        const int outputLen,
        const int axis,
        const int numTensors,
        const int *shapeAtAxis)
    {
        auto [numBlocks, blockSize] = BestDims(outputLen);

        // output
        cudaVector<int> outputStrides_d(outputStrides, dim, hipMemcpyHostToDevice);

        // input
        // datas
        cudaVector<const T *> tensorsDataList(tensorsData, numTensors, hipMemcpyHostToDevice);
        // strides
        cudaVector<int> inputStrides_d(inputStrides, numTensors * dim, hipMemcpyHostToDevice);

        // shapeAtAxis
        cudaVector<int> shapeAtAxis_d(shapeAtAxis, numTensors, hipMemcpyHostToDevice);
        switch (dim)
        {
        case 1:
            concat_kernel<1, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 2:
            concat_kernel<2, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 3:
            concat_kernel<3, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 4:
            concat_kernel<4, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 5:
            concat_kernel<5, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 6:
            concat_kernel<6, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 7:
            concat_kernel<7, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 8:
            concat_kernel<8, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 9:
            concat_kernel<9, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 10:
            concat_kernel<10, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 11:
            concat_kernel<11, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;
        case 12:
            concat_kernel<12, T><<<numBlocks, blockSize>>>(tensorsDataList.data, inputStrides_d.data, outputData, outputStrides_d.data, dim, outputLen, axis, numTensors, shapeAtAxis_d.data);
            break;

        default:
            throw std::runtime_error("dimension large than " + std::to_string(MAX_DIM));
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("cuda error");
        }
    }
    template void launch_concat<double>(const double **tensorsData, const int *inputStrides, double *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);
    template void launch_concat<float>(const float **tensorsData, const int *inputStrides, float *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);
    template void launch_concat<hip_bfloat16>(const hip_bfloat16 **tensorsData, const int *inputStrides, hip_bfloat16 *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);
    template void launch_concat<__half>(const __half **tensorsData, const int *inputStrides, __half *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);
    template void launch_concat<int64_t>(const int64_t **tensorsData, const int *inputStrides, int64_t *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);
    template void launch_concat<int32_t>(const int32_t **tensorsData, const int *inputStrides, int32_t *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);
    template void launch_concat<int16_t>(const int16_t **tensorsData, const int *inputStrides, int16_t *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);
    template void launch_concat<int8_t>(const int8_t **tensorsData, const int *inputStrides, int8_t *outputData, const int *outputStrides, const int dim, const int len, const int axis, const int numTensors, const int *shapeAtAxis);

    // broadcastTo
    __host__ __device__ void fromBroadcastIndices(const BroadcastMap *broadcastMap, const int *broadcastIndices, const int broadcastIndicesDim, int *indices)
    {
        for (int i = 0, j = 0; i < broadcastIndicesDim; ++i)
        {
            switch (broadcastMap[i])
            {
            case xTox:
                indices[j++] = broadcastIndices[i];
                break;
            case nullTo1:
                break;
            case xTo1:
                indices[j++] = 0;
                break;
            }
        }
    }

    template <int DIM, typename T>
    __global__ void broadcastTo_kernel(const T *input, const int *inputStrides, const int inputDim,
                                       const BroadcastMap *broadcastMap,
                                       T *output, const int *outputStrides, const int outputDim, const int outputlen)
    {
        const int grid_stride = gridDim.x * blockDim.x;
        int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
        for (; thread_id < outputlen; thread_id += grid_stride)
        {
            int output_indices[DIM];
            linearTo(outputStrides, outputDim, output_indices, thread_id);
            int input_indices[DIM];
            fromBroadcastIndices(broadcastMap, output_indices, outputDim, input_indices);
            int inputIdx = linearAt(inputStrides, inputDim, input_indices);
            int outputIdx = linearAt(outputStrides, outputDim, output_indices);
            output[outputIdx] = input[inputIdx];
        }
    }

    template <typename T>
    void launch_broadcastTo(const T *input, const int *inputStrides, const int intputDim,
                            const BroadcastMap *broadcastMap,
                            T *output, const int *outputStrides, const int outputDim, const int outputlen)
    {

        auto [numBlocks, blockSize] = BestDims(outputlen);

        // output
        cudaVector<int> outputStrides_d(outputStrides, outputDim, hipMemcpyHostToDevice);

        // broadcastMap
        cudaVector<BroadcastMap> broadcastMap_d(broadcastMap, outputDim, hipMemcpyHostToDevice);

        // input
        cudaVector<int> inputStrides_d(inputStrides, intputDim, hipMemcpyHostToDevice);

        switch (outputDim)
        {
        case 1:
            broadcastTo_kernel<1, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 2:
            broadcastTo_kernel<2, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 3:
            broadcastTo_kernel<3, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 4:
            broadcastTo_kernel<4, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 5:
            broadcastTo_kernel<5, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 6:
            broadcastTo_kernel<6, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 7:
            broadcastTo_kernel<7, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 8:
            broadcastTo_kernel<8, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 9:
            broadcastTo_kernel<9, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 10:
            broadcastTo_kernel<10, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 11:
            broadcastTo_kernel<11, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 12:
            broadcastTo_kernel<12, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, intputDim, broadcastMap_d.data, output, outputStrides_d.data, outputDim, outputlen);
            break;
        default:
            throw std::runtime_error("dimension large than " + std::to_string(MAX_DIM));
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("cuda error");
        }
    }
    template void launch_broadcastTo<double>(const double *input, const int *inputStrides, const int inputDim,
                                             const BroadcastMap *broadcastMap,
                                             double *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_broadcastTo<float>(const float *input, const int *inputStrides, const int inputDim,
                                            const BroadcastMap *broadcastMap,
                                            float *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_broadcastTo<hip_bfloat16>(const hip_bfloat16 *input, const int *inputStrides, const int inputDim,
                                                  const BroadcastMap *broadcastMap,
                                                  hip_bfloat16 *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_broadcastTo<__half>(const __half *input, const int *inputStrides, const int inputDim,
                                             const BroadcastMap *broadcastMap,
                                             __half *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_broadcastTo<int64_t>(const int64_t *input, const int *inputStrides, const int inputDim,
                                              const BroadcastMap *broadcastMap,
                                              int64_t *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_broadcastTo<int32_t>(const int32_t *input, const int *inputStrides, const int inputDim,
                                              const BroadcastMap *broadcastMap,
                                              int32_t *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_broadcastTo<int16_t>(const int16_t *input, const int *inputStrides, const int inputDim,
                                              const BroadcastMap *broadcastMap,
                                              int16_t *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_broadcastTo<int8_t>(const int8_t *input, const int *inputStrides, const int inputDim,
                                             const BroadcastMap *broadcastMap,
                                             int8_t *output, const int *outputStrides, const int outputDim, const int outputlen);

    // gather

    __host__ __device__ void fromGatherIndices(
        const int *output_indices, const int outputDim,
        const int *indices,
        const int gatherAxis,
        int *input_indices)
    {
        // 1. 首先复制所有输出索引到输入索引
        for (int i = 0; i < outputDim; i++)
        {
            input_indices[i] = output_indices[i];
        }

        // 2. 使用indices中存储的值作为gatherAxis维度的索引
        // 这里假设indices是一个一维数组，索引位置由output_indices[gatherAxis]确定
        input_indices[gatherAxis] = indices[output_indices[gatherAxis]];
    }

    template <int DIM, typename T>
    __global__ void gather_kernel(const T *input, const int *inputStrides, const int inputDim,
                                  const int *indices, const int *indicesStrides, const int indicesDim,
                                  const int gatherAxis,
                                  T *output, const int *outputStrides, const int outputDim, const int outputlen)
    {
        const int grid_stride = gridDim.x * blockDim.x;
        int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
        for (; thread_id < outputlen; thread_id += grid_stride)
        {
            // 输出索引
            int output_indices[DIM];
            linearTo(outputStrides, outputDim, output_indices, thread_id);

            // 输入索引
            int input_indices[DIM];
            fromGatherIndices(indices_indices, gatherAxis, input_indices);
            int inputIdx = linearAt(inputStrides, inputDim, input_indices);
            int outputIdx = linearAt(outputStrides, outputDim, output_indices);
            output[outputIdx] = input[inputIdx];
        }
    }

    template <typename T>
    void launch_gather(const T *input, const int *inputStrides, const int inputDim,
                       const int *indices, const int *indicesStrides, const int indicesDim,
                       const int gatherAxis,
                       T *output, const int *outputStrides, const int outputDim, const int outputlen)
    {

        auto [numBlocks, blockSize] = BestDims(outputlen);

        // output
        cudaVector<int> outputStrides_d(outputStrides, outputDim, hipMemcpyHostToDevice);

        // indices
        cudaVector<int> indicesStrides_d(indicesStrides, indicesDim, hipMemcpyHostToDevice);

        // input
        cudaVector<int> inputStrides_d(inputStrides, inputDim, hipMemcpyHostToDevice);

        switch (outputDim)
        {
        case 1:
            gather_kernel<1, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 2:
            gather_kernel<2, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 3:
            gather_kernel<3, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 4:
            gather_kernel<4, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 5:
            gather_kernel<5, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 6:
            gather_kernel<6, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 7:
            gather_kernel<7, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 8:
            gather_kernel<8, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 9:
            gather_kernel<9, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 10:
            gather_kernel<10, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 11:
            gather_kernel<11, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        case 12:
            gather_kernel<12, T><<<numBlocks, blockSize>>>(input, inputStrides_d.data, inputDim, indices_d.data, indicesStrides_d.data, indicesDim, gatherAxis, output, outputStrides_d.data, outputDim, outputlen);
            break;
        default:
            throw std::runtime_error("dimension large than " + std::to_string(MAX_DIM));
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("cuda error");
        }
    }
    template void launch_gather<double>(const double *input, const int *inputStrides, const int inputDim,
                                        const int *indices, const int *indicesStrides, const int indicesDim,
                                        const int gatherAxis,
                                        double *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<float>(const float *input, const int *inputStrides, const int inputDim,
                                       const int *indices, const int *indicesStrides, const int indicesDim,
                                       const int gatherAxis,
                                       float *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<hip_bfloat16>(const hip_bfloat16 *input, const int *inputStrides, const int inputDim,
                                             const int *indices, const int *indicesStrides, const int indicesDim,
                                             const int gatherAxis,
                                             hip_bfloat16 *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<__half>(const __half *input, const int *inputStrides, const int inputDim,
                                        const int *indices, const int *indicesStrides, const int indicesDim,
                                        const int gatherAxis,
                                        __half *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<int64_t>(const int64_t *input, const int *inputStrides, const int inputDim,
                                         const int *indices, const int *indicesStrides, const int indicesDim,
                                         const int gatherAxis,
                                         int64_t *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<int32_t>(const int32_t *input, const int *inputStrides, const int inputDim,
                                         const int *indices, const int *indicesStrides, const int indicesDim,
                                         const int gatherAxis,
                                         int32_t *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<int16_t>(const int16_t *input, const int *inputStrides, const int inputDim,
                                         const int *indices, const int *indicesStrides, const int indicesDim,
                                         const int gatherAxis,
                                         int16_t *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<int8_t>(const int8_t *input, const int *inputStrides, const int inputDim,
                                        const int *indices, const int *indicesStrides, const int indicesDim,
                                        const int gatherAxis,
                                        int8_t *output, const int *outputStrides, const int outputDim, const int outputlen);
    template void launch_gather<bool>(const bool *input, const int *inputStrides, const int inputDim,
                                      const int *indices, const int *indicesStrides, const int indicesDim,
                                      const int gatherAxis,
                                      bool *output, const int *outputStrides, const int outputDim, const int outputlen);

};
#endif // DEEPX_TENSORFUNC_CHANGESHAPE_MIAOBYTE_HPP