#include "hip/hip_runtime.h"
#ifndef DEEPX_TENSORFUNC_ELEMENTWISE_MIAOBYTE_SIN_CU
#define DEEPX_TENSORFUNC_ELEMENTWISE_MIAOBYTE_SIN_CU

#include <hip/hip_fp16.h>

 
#include "deepx/tensorfunc/cuda.hpp"
#include "deepx/tensorfunc/authors.hpp"

namespace deepx::tensorfunc
{
    // sin
    template <typename T>
    __global__ void sin_kernel(const T* A, T* C, const int size);
    
    template <>
    __global__ void sin_kernel<double>(const double* A, double* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = sin(A[idx]);
        }
    }
    template <>
    __global__ void sin_kernel<float>(const float* A, float* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = sinf(A[idx]);
        }
    }

    template <>
    __global__ void sin_kernel<__half>(const __half* A, __half* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = hsin(A[idx]);
        }
    }   
 
    template <typename T>
    void launch_sin(int numBlocks, int blockSize, const T* a, T* c, const int size){
        sin_kernel<<<numBlocks, blockSize>>>(a, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch sin kernel: " + 
                                       std::string(hipGetErrorString(err)));
        }
    }

    template void  launch_sin<double>(int numBlocks, int blockSize, const double* a, double* c, const int size);
    template void  launch_sin<float>(int numBlocks, int blockSize, const float* a, float* c, const int size);
    template void  launch_sin<__half>(int numBlocks, int blockSize, const __half* a, __half* c, const int size);

    // cos
    template <typename T>
    __global__ void cos_kernel(const T* A, T* C, const int size);
    template <>
    __global__ void cos_kernel<double>(const double* A, double* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = cos(A[idx]);
        }
    }
    template <>
    __global__ void cos_kernel<float>(const float* A, float* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = cosf(A[idx]);
        }
    }   
 
    template <>
    __global__ void cos_kernel<__half>(const __half* A, __half* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = hcos(A[idx]);
        }
    }      
 
    template <typename T>
    void launch_cos(int numBlocks, int blockSize, const T* a, T* c, const int size){
        cos_kernel<<<numBlocks, blockSize>>>(a, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch cos kernel: " + 
                                       std::string(hipGetErrorString(err)));
        }
    }
    template void  launch_cos<double>(int numBlocks, int blockSize, const double* a, double* c, const int size);    
    template void  launch_cos<float>(int numBlocks, int blockSize, const float* a, float* c, const int size);
    template void  launch_cos<__half>(int numBlocks, int blockSize, const __half* a, __half* c, const int size);
 
    // tan
    template <typename T>
    __global__ void tan_kernel(const T* A, T* C, const int size);
    template <>
    __global__ void tan_kernel<double>(const double* A, double* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = tan(A[idx]);
        }
    }   
    template <>
    __global__ void tan_kernel<float>(const float* A, float* C, const int size){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = tanf(A[idx]);
        }
    }
   
 
    template <typename T>   
    void launch_tan(int numBlocks, int blockSize, const T* a, T* c, const int size){
        tan_kernel<<<numBlocks, blockSize>>>(a, c, size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to launch tan kernel: " + 
                                       std::string(hipGetErrorString(err)));
        }
    }
    template void  launch_tan<double>(int numBlocks, int blockSize, const double* a, double* c, const int size);
    template void  launch_tan<float>(int numBlocks, int blockSize, const float* a, float* c, const int size);
 
}

#endif // DEEPX_TENSORFUNC_ELEMENTWISE_MIAOBYTE_SIN_CU
