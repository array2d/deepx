#include "hip/hip_runtime.h"
#include "deepx/tensorfunc/cuda.hpp"

#include <cuda_fp64.h>
#include <cuda_fp32.h>
#include "deepx/tensor.hpp"

#include "deepx/tensorfunc/matmul.hpp"
#include "deepx/tensorfunc/authors.hpp"
#include "deepx/tensorfunc/cuda.hpp"

namespace deepx::tensorfunc {

  #define BLOCK_SIZE 32

  __global__ void fp64MatmulKernel(double *C, const double *A, const double *B, 
                                   int M, int N, int K) {
      // 定义共享内存块，用于缓存A和B的矩阵块
      __shared__ double tileA[BLOCK_SIZE][BLOCK_SIZE];
      __shared__ double tileB[BLOCK_SIZE][BLOCK_SIZE];
  
      // 计算当前线程处理的全局矩阵位置
      int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
      int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  
      double sum = 0.0;
  
      // 分块循环处理整个K维度
      for (int t = 0; t < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
          // 计算当前块的起始位置
          int tiledK = t * BLOCK_SIZE;
  
          // 加载A的块到共享内存（行优先）
          int loadA_col = tiledK + threadIdx.x;
          if (row < M && loadA_col < K) {
              tileA[threadIdx.y][threadIdx.x] = A[row * K + loadA_col];
          } else {
              tileA[threadIdx.y][threadIdx.x] = 0.0;  // 填充0处理边界
          }
  
          // 加载B的块到共享内存（列优先等效处理）
          int loadB_row = tiledK + threadIdx.y;
          if (col < N && loadB_row < K) {
              tileB[threadIdx.y][threadIdx.x] = B[loadB_row * N + col];
          } else {
              tileB[threadIdx.y][threadIdx.x] = 0.0;  // 填充0处理边界
          }
  
          __syncthreads();  // 确保块加载完成
  
          // 计算当前块的矩阵乘法贡献
          for (int k = 0; k < BLOCK_SIZE; ++k) {
              sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
          }
  
          __syncthreads();  // 确保计算完成再加载下一块
      }
  
      // 只将有效范围内的结果写入全局内存
      if (row < M && col < N) {
          C[row * N + col] = sum;
      }
  }
  
  // 主机函数调用内核
  void fp64Matmul(double *d_C, const double *d_A, const double *d_B, 
                  int M, int N, int K) {
      dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
      dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                   (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
      
      fp64MatmulKernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, M, N, K);
  }
  

}

} // namespace tensorfunc
} // namespace deepx
